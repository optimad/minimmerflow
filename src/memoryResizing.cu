/*---------------------------------------------------------------------------*\
 *
 *  minimmerflow
 *
 *  Copyright (C) 2015-2021 OPTIMAD engineering Srl
 *
 *  -------------------------------------------------------------------------
 *  License
 *  This file is part of minimmerflow.
 *
 *  minimmerflow is free software: you can redistribute it and/or modify it
 *  under the terms of the GNU Lesser General Public License v3 (LGPL)
 *  as published by the Free Software Foundation.
 *
 *  minimmerflow is distributed in the hope that it will be useful, but WITHOUT
 *  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 *  FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public
 *  License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with minimmerflow. If not, see <http://www.gnu.org/licenses/>.
 *
\*---------------------------------------------------------------------------*/

#ifndef __MINIMMEFLOW_ADAPTATIONMANAGER_CU__
#define __MINIMMEFLOW_ADAPTATIONMANAGER_CU__

#include "memoryResizing.hcu"
#include <hip/hip_runtime.h>


/*!
 * Constructor
 */
MemoryResizing::MemoryResizing()
:
    m_dp(nullptr), m_prop(), m_handles(), m_allocSize(0ULL), m_reservedSize(0ULL), m_chunkSize(0ULL)
{
    hipError_t status = hipSuccess;
    // TODO: This is where the cuda-gdb hangs:
    status = hipInit(0);
    CHECK_DRV(status);
    int device;
    (void)status;

    CUDA_ERROR_CHECK(hipGetDevice(&device));
    assert(status == hipSuccess);

    m_prop.type = hipMemAllocationTypePinned;
    m_prop.location.type = hipMemLocationTypeDevice;
    m_prop.location.id = (int)device;

    m_accessDesc.location = m_prop.location;
    m_accessDesc.flags = hipMemAccessFlagsProtReadWrite;

    status = hipMemGetAllocationGranularity(&m_chunkSize, &m_prop, hipMemAllocationGranularityMinimum);
    CHECK_DRV(status);
    assert(status == hipSuccess);
}


/*!
 * Destructor
 */
MemoryResizing::~MemoryResizing()
{
    hipError_t status = hipSuccess;
    (void)status;
    if (m_dp != nullptr) {
        if (*m_dp != 0ULL) {
            status = hipMemUnmap(*m_dp, m_allocSize);
            CHECK_DRV(status);
            assert(status == hipSuccess);
            for (size_t i = 0; i < m_vaRanges.size(); i++) {
                status = hipMemAddressFree(m_vaRanges[i].start, m_vaRanges[i].sz);
                CHECK_DRV(status);
                assert(status == hipSuccess);
            }
            for (size_t i = 0ULL; i < m_handles.size(); i++) {
                status = hipMemRelease(m_handles[i]);
                CHECK_DRV(status);
                assert(status == hipSuccess);
            }
        }
    }
}


/*!
 * Reserve virtual memory of array on GPU
 */
hipError_t MemoryResizing::cuda_reserve(size_t new_sz)
{
    hipError_t status = hipSuccess;

    hipDeviceptr_t new_ptr = 0ULL;

    if (new_sz <= m_reservedSize) {
        return hipSuccess;
    }

    const size_t aligned_sz = ((new_sz + m_chunkSize - 1) / m_chunkSize) * m_chunkSize;

    status = hipMemAddressReserve(&new_ptr, (aligned_sz - m_reservedSize), 0ULL, *m_dp + m_reservedSize, 0ULL);
    CHECK_DRV(status);

    // Try to reserve an address just after what we already have reserved
    if (status != hipSuccess || (new_ptr != *m_dp + m_reservedSize)) {
        if (new_ptr != 0ULL) {
            (void)hipMemAddressFree(new_ptr, (aligned_sz - m_reservedSize));
        }
        // Slow path - try to find a new address reservation big enough for us
        status = hipMemAddressReserve(&new_ptr, aligned_sz, 0ULL, 0U, 0);
        CHECK_DRV(status);
        if (status == hipSuccess && *m_dp != 0ULL) {
            hipDeviceptr_t ptr = new_ptr;
            // Found one, now unmap our previous allocations
            status = hipMemUnmap(*m_dp, m_allocSize);
            CHECK_DRV(status);
            assert(status == hipSuccess);
            for (size_t i = 0ULL; i < m_handles.size(); i++) {
                const size_t hdl_sz = m_handleSizes[i];
                // And remap them, enabling their access
                status = hipMemMap(ptr, hdl_sz, 0ULL, m_handles[i], 0ULL);
                CHECK_DRV(status);
                if (status != hipSuccess)
                    break;
                status = hipMemSetAccess(ptr, hdl_sz, &m_accessDesc, 1ULL);
                CHECK_DRV(status);
                if (status != hipSuccess)
                    break;
                ptr += hdl_sz;
            }
            if (status != hipSuccess) {
                // Failed the mapping somehow... clean up!
                status = hipMemUnmap(new_ptr, aligned_sz);
                CHECK_DRV(status);
                assert(status == hipSuccess);
                status = hipMemAddressFree(new_ptr, aligned_sz);
                CHECK_DRV(status);
                assert(status == hipSuccess);
            }
            else {
                // Clean up our old VA reservations!
                for (size_t i = 0ULL; i < m_vaRanges.size(); i++) {
                    (void)hipMemAddressFree(m_vaRanges[i].start, m_vaRanges[i].sz);
                }
                m_vaRanges.clear();
            }
        }
        // Assuming everything went well, update everything
        if (status == hipSuccess) {
            *m_dp = new_ptr;
            m_reservedSize = aligned_sz;
            cuda_addVARange(new_ptr, aligned_sz);
        }
    }
    else {
        cuda_addVARange(new_ptr, aligned_sz - m_reservedSize);
        if (*m_dp == 0ULL) {
            *m_dp = new_ptr;
        }
        m_reservedSize = aligned_sz;
    }

    return status;
}


/*!
 * Grow array on GPU
 */
hipError_t MemoryResizing::cuda_grow(std::size_t new_sz)
{
    hipError_t status = hipSuccess;
    hipMemGenericAllocationHandle_t handle;
    if (new_sz <= m_allocSize) {
        return hipSuccess;
    }

    const size_t size_diff = new_sz - m_allocSize;
    // Round up to the next chunk size
    const size_t sz = ((size_diff + m_chunkSize - 1) / m_chunkSize) * m_chunkSize;
    status = cuda_reserve(m_allocSize + sz);
    CHECK_DRV(status);

    if (status != hipSuccess) {
        return status;
    }
    status = hipMemCreate(&handle, sz, &m_prop, 0);
    CHECK_DRV(status);
    if (status == hipSuccess) {
        status = hipMemMap(*m_dp + m_allocSize, sz, 0ULL, handle, 0ULL);
        CHECK_DRV(status);
        if (status == hipSuccess) {
            status = hipMemSetAccess(*m_dp + m_allocSize, sz, &m_accessDesc, 1ULL);
            CHECK_DRV(status);
            if (status == hipSuccess) {
                m_handles.push_back(handle);
                m_handleSizes.push_back(sz);
                m_allocSize += sz;
            }
            if (status != hipSuccess) {
                (void)hipMemUnmap(*m_dp + m_allocSize, sz);
            }
        }
        if (status != hipSuccess) {
            (void)hipMemRelease(handle);
        }
    }
    return status;
}


#endif
