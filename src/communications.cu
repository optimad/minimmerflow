/*---------------------------------------------------------------------------*\
 *
 *  minimmerflow
 *
 *  Copyright (C) 2015-2022 OPTIMAD engineering Srl
 *
 *  -------------------------------------------------------------------------
 *  License
 *  This file is part of minimmerflow.
 *
 *  minimmerflow is free software: you can redistribute it and/or modify it
 *  under the terms of the GNU Lesser General Public License v3 (LGPL)
 *  as published by the Free Software Foundation.
 *
 *  minimmerflow is distributed in the hope that it will be useful, but WITHOUT
 *  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 *  FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public
 *  License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with minimmerflow. If not, see <http://www.gnu.org/licenses/>.
 *
\*---------------------------------------------------------------------------*/

#include "communications.hcu"
#include "communications.hpp"
#include "containers.hcu"

// Include template implementation
#include "communications.tcu"

#include "hip/hip_runtime_api.h"

// Explicit instantiation

template class CudaStorageBufferStreamer<std::unordered_map<int, ScalarStorage<double>>>;
template class CudaStorageCollectionBufferStreamer<std::unordered_map<int, ScalarStorage<double>>>;


/*!
 * Send ghosts data using non-blocking communications
 *
 * \param cellData is the container of the cell data
 */
void ListCommunicator::startAllExchanges()
{
    if (getCommunicator() == MPI_COMM_NULL || !hasData()) {
        return;
    }

    // Start the receives
    for (int rank : getRecvRanks()) {
        if (!isRecvActive(rank)) {
            startRecv(rank);
        }
    }

    // Wait previous sends
    waitAllSends();

    // Fill the buffer with the given field and start sending the data
    //WARNING: the follow part works if only one writer has been added to the communicator
    CudaStorageCollectionBufferStreamer<std::unordered_map<int, ScalarStorage<double>>> * cudaStreamer =
            static_cast<CudaStorageCollectionBufferStreamer<std::unordered_map<int, ScalarStorage<double>>> *>(m_writers[0]);
    for (int rank : getSendRanks()) {
        bitpit::SendBuffer &buffer = getSendBuffer(rank);

        cudaStreamer->prepareWrite(rank, buffer, getStreamableSendList(rank, cudaStreamer));
        cudaStreamer->write(rank, buffer, getStreamableSendList(rank, cudaStreamer));
    }
    for (int rank : getSendRanks()) {
        hipStreamSynchronize(cudaStreamer->m_queuesStreams.getCudaStreamByRank(rank));
        startSend(rank);
    }

}

void ListCommunicator::prepare1StartAllExchanges()
{
    if (getCommunicator() == MPI_COMM_NULL || !hasData()) {
        return;
    }

    // Start the receives
    for (int rank : getRecvRanks()) {
        if (!isRecvActive(rank)) {
            startRecv(rank);
        }
    }

    // Wait previous sends
    waitAllSends();

    // Fill the buffer with the given field and start sending the data
    //WARNING: the follow part works if only one writer has been added to the communicator
    CudaStorageCollectionBufferStreamer<std::unordered_map<int, ScalarStorage<double>>> * cudaStreamer =
            static_cast<CudaStorageCollectionBufferStreamer<std::unordered_map<int, ScalarStorage<double>>> *>(m_writers[0]);
    for (int rank : getSendRanks()) {
        bitpit::SendBuffer &buffer = getSendBuffer(rank);

        cudaStreamer->prepareWrite(rank, buffer, getStreamableSendList(rank, cudaStreamer));
    }
}

void ListCommunicator::prepare2StartAllExchanges()
{
    CudaStorageCollectionBufferStreamer<std::unordered_map<int, ScalarStorage<double>>> * cudaStreamer =
            static_cast<CudaStorageCollectionBufferStreamer<std::unordered_map<int, ScalarStorage<double>>> *>(m_writers[0]);
    for (int rank : getSendRanks()) {
        bitpit::SendBuffer &buffer = getSendBuffer(rank);

        cudaStreamer->write(rank, buffer, getStreamableSendList(rank, cudaStreamer));
    }
}

void ListCommunicator::completeStartAllExchanges()
{
    CudaStorageCollectionBufferStreamer<std::unordered_map<int, ScalarStorage<double>>> * cudaStreamer =
            static_cast<CudaStorageCollectionBufferStreamer<std::unordered_map<int, ScalarStorage<double>>> *>(m_writers[0]);
    for (int rank : getSendRanks()) {
        hipStreamSynchronize(cudaStreamer->m_queuesStreams.getCudaStreamByRank(rank));
        startSend(rank);
    }
}
void ListCommunicator::initializeCudaObjects()
{
    if (getCommunicator() == MPI_COMM_NULL || !hasData()) {
        return;
    }
    for (int rank : getSendRanks()) {
        bitpit::SendBuffer &buffer = getSendBuffer(rank);
        std::size_t bufferSize = buffer.getSize();
        size_t bytes = bufferSize * sizeof(char);
        hipError_t err = hipHostRegister(buffer.getFront().data(), bytes, hipHostRegisterDefault);
        if (err != hipSuccess) {
            std::cout << "CUDA runtime error in hipHostRegister " << hipGetErrorString(err) << " on buffer for rank " << rank << std::endl;
        }
    }
    for (int rank : getRecvRanks()) {
        bitpit::RecvBuffer &buffer = getRecvBuffer(rank);
        std::size_t bufferSize = buffer.getSize();
        size_t bytes = bufferSize * sizeof(char);
        hipError_t err = hipHostRegister(buffer.getFront().data(), bytes, hipHostRegisterDefault);
        if (err != hipSuccess) {
            std::cout << "CUDA runtime error in hipHostRegister " << hipGetErrorString(err) << " on buffer for rank " << rank << std::endl;
        }
    }

}
void ListCommunicator::finalizeCudaObjects()
{
    if (getCommunicator() == MPI_COMM_NULL || !hasData()) {
        return;
    }
    for (int rank : getSendRanks()) {
        bitpit::SendBuffer &buffer = getSendBuffer(rank);
        hipError_t err = hipHostUnregister(buffer.getFront().data());
        if (err != hipSuccess) {
            std::cout << "CUDA runtime error in hipHostUnregister (SEND)" << hipGetErrorString(err)  << " pointer " << (void *)buffer.getFront().data()
                    << " Comm " << m_name << std::endl;
        }
    }
    for (int rank : getRecvRanks()) {
        bitpit::RecvBuffer &buffer = getRecvBuffer(rank);
        hipError_t err = hipHostUnregister(buffer.getFront().data());
        if (err != hipSuccess) {
            std::cout << "CUDA runtime error in hipHostUnregister (RECV)" << hipGetErrorString(err)  << " pointer " << (void *)buffer.getFront().data()
                    << " Comm " << m_name << std::endl;
        }
    }
}


ListCommunicator::~ListCommunicator()
{
}

QueuesStreams::QueuesStreams(const std::unordered_map<int, std::vector<long>> & sourceLists,
        const std::unordered_map<int, std::vector<long>> & targetLists, MPI_Comm communicator)
{
    std::set<int> sourceRanks, targetRanks;
    for (const auto & sourceList :sourceLists) {
        sourceRanks.insert(sourceList.first);
    }
    for (const auto & targetList :targetLists) {
        targetRanks.insert(targetList.first);
    }
    bool debug = sourceRanks == targetRanks;
    MPI_Allreduce(MPI_IN_PLACE, &debug, 1, MPI_CXX_BOOL, MPI_LAND, communicator);

    if (debug) {
        std::runtime_error("========================================================> Source ranks different from Target ranks! Cannot continue...");
    }


    for (int r : sourceRanks) {
        m_queueIds[r] = r+101;
        hipStreamCreate(&(m_cudaStreams[r]));
        acc_set_cuda_stream(m_queueIds[r], m_cudaStreams[r]);
    }

    int rank;
    MPI_Comm_rank(communicator, &rank);
    for (const auto & queue : m_queueIds) {
        std::cout << "Rank " << rank << " " << queue.first << "/" << queue.second << std::endl;
    }
}

QueuesStreams::~QueuesStreams()
{
    for (int i = 0; i < m_cudaStreams.size(); ++i) {
        hipStreamDestroy(m_cudaStreams[i]);
    }
}

hipStream_t & QueuesStreams::getCudaStreamByRank(int rank)
{
    return m_cudaStreams.at(rank);
}
int & QueuesStreams::getOpenACCQueueByRank(int rank)
{
    return m_queueIds.at(rank);
}
