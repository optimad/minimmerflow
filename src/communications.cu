/*---------------------------------------------------------------------------*\
 *
 *  minimmerflow
 *
 *  Copyright (C) 2015-2022 OPTIMAD engineering Srl
 *
 *  -------------------------------------------------------------------------
 *  License
 *  This file is part of minimmerflow.
 *
 *  minimmerflow is free software: you can redistribute it and/or modify it
 *  under the terms of the GNU Lesser General Public License v3 (LGPL)
 *  as published by the Free Software Foundation.
 *
 *  minimmerflow is distributed in the hope that it will be useful, but WITHOUT
 *  ANY WARRANTY; without even the implied warranty of MERCHANTABILITY or
 *  FITNESS FOR A PARTICULAR PURPOSE. See the GNU Lesser General Public
 *  License for more details.
 *
 *  You should have received a copy of the GNU Lesser General Public License
 *  along with minimmerflow. If not, see <http://www.gnu.org/licenses/>.
 *
\*---------------------------------------------------------------------------*/


#include "communications.hcu"
#include "communications.hpp"
#include "containers.hcu"

// Include template implementation
#include "communications.tcu"

#include "hip/hip_runtime_api.h"

// Explicit instantiation

template class CudaStorageBufferStreamer<std::unordered_map<int, ScalarStorage<double>>>;
template class CudaStorageCollectionBufferStreamer<std::unordered_map<int, ScalarStorage<double>>>;


/*!
 * Send ghosts data using non-blocking communications
 *
 * \param cellData is the container of the cell data
 */
void ListCommunicator::startAllExchanges()
{
    if (getCommunicator() == MPI_COMM_NULL || !hasData()) {
        return;
    }

    // Start the receives
    for (int rank : getRecvRanks()) {
        if (!isRecvActive(rank)) {
            startRecv(rank);
        }
    }

    // Wait previous sends
    waitAllSends();

    // Fill the buffer with the given field and start sending the data
    //WARNING: the follow part works if only one writer has been added to the communicator
    CudaStorageCollectionBufferStreamer<std::unordered_map<int, ScalarStorage<double>>> * cudaStreamer =
            static_cast<CudaStorageCollectionBufferStreamer<std::unordered_map<int, ScalarStorage<double>>> *>(m_writers[0]);
    for (int rank : getSendRanks()) {
        bitpit::SendBuffer &buffer = getSendBuffer(rank);

        cudaStreamer->prepareWrite(rank, buffer, getStreamableSendList(rank, cudaStreamer));
        cudaStreamer->write(rank, buffer, getStreamableSendList(rank, cudaStreamer));
    }
    for (int rank : getSendRanks()) {
//        hipStreamSynchronize(cudaStreamer->m_cudaStreams[rank]);
        startSend(rank);
    }

}

void ListCommunicator::initializeCudaObjects()
{
    if (getCommunicator() == MPI_COMM_NULL || !hasData()) {
        return;
    }
    for (int rank : getSendRanks()) {
        bitpit::SendBuffer &buffer = getSendBuffer(rank);
        std::size_t bufferSize = buffer.getSize();
        size_t bytes = bufferSize * sizeof(char);
        hipError_t err = hipHostRegister(buffer.getFront().data(), bytes, hipHostRegisterDefault);
        if (err != hipSuccess) {
            std::cout << "CUDA runtime error in hipHostRegister " << hipGetErrorString(err) << " on buffer for rank " << rank << std::endl;
        }
    }
    for (int rank : getRecvRanks()) {
        bitpit::RecvBuffer &buffer = getRecvBuffer(rank);
        std::size_t bufferSize = buffer.getSize();
        size_t bytes = bufferSize * sizeof(char);
        hipError_t err = hipHostRegister(buffer.getFront().data(), bytes, hipHostRegisterDefault);
        if (err != hipSuccess) {
            std::cout << "CUDA runtime error in hipHostRegister " << hipGetErrorString(err) << " on buffer for rank " << rank << std::endl;
        }
    }

}
void ListCommunicator::finalizeCudaObjects()
{
    if (getCommunicator() == MPI_COMM_NULL || !hasData()) {
        return;
    }
    for (int rank : getSendRanks()) {
        bitpit::SendBuffer &buffer = getSendBuffer(rank);
        hipError_t err = hipHostUnregister(buffer.getFront().data());
        if (err != hipSuccess) {
            std::cout << "CUDA runtime error in hipHostUnregister (SEND)" << hipGetErrorString(err)  << " pointer " << (void *)buffer.getFront().data()
                    << " Comm " << m_name << std::endl;
        }
    }
    for (int rank : getRecvRanks()) {
        bitpit::RecvBuffer &buffer = getRecvBuffer(rank);
        hipError_t err = hipHostUnregister(buffer.getFront().data());
        if (err != hipSuccess) {
            std::cout << "CUDA runtime error in hipHostUnregister (RECV)" << hipGetErrorString(err)  << " pointer " << (void *)buffer.getFront().data()
                    << " Comm " << m_name << std::endl;
        }
    }
}


ListCommunicator::~ListCommunicator()
{
}

OpenACCStreams::OpenACCStreams(int nFields)
{
    m_cudaStreams.resize(nFields);
    m_streamIds.resize(nFields, 0);
    for (int i = 0; i < nFields; ++i) {
        m_streamIds[i] = i;
        hipStreamCreate(&(m_cudaStreams[i]));
        acc_set_cuda_stream(m_streamIds[i], m_cudaStreams[i]);
    }
}
OpenACCStreams::~OpenACCStreams()
{
    for (int i = 0; i < m_cudaStreams.size(); ++i) {
        hipStreamDestroy(m_cudaStreams[i]);
    }
}
