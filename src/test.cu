#include "hip/hip_runtime.h"
#include "test.cuh"

__global__
void add(int n, double *x, double *y, double *z)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;
    z[i] = x[i] + y[i];
}

namespace CudaWrappers{

    void  add_wrapper(int N, double *x, double *y, double *z, int blockSize) {
    
        // Allocate on GPU
        double *dx, *dy, *dz;
        int size = N * sizeof(double);
    
        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        hipMalloc((void **) &dx, size);
        hipMalloc((void **) &dy, size);
        hipMalloc((void **) &dz, size);
        hipMemcpy(dx, x, size, hipMemcpyHostToDevice);
        hipMemcpy(dy, y, size, hipMemcpyHostToDevice);
        hipMemcpy(dz, z, size, hipMemcpyHostToDevice);

    
        // Run kernel on 1M elements on the GPU
        int numBlocks = (N + blockSize - 1) / blockSize;
        add<<<numBlocks, blockSize>>>(N, dx, dy, dz);

        hipError_t mycudaerror = hipGetLastError() ;
    
        hipMemcpy(z, dz, N*sizeof(double), hipMemcpyDeviceToHost);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
       
        hipFree(dx);
        hipFree(dy);
        hipFree(dz);

    }

}
